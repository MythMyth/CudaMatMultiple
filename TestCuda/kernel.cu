﻿
#include "hip/hip_runtime.h"


#include "stdio.h"

hipError_t matMulWithCuda(int* c, const int* a, const int* b);

constexpr int TILE_SIZE = 16;

__global__ void MatMul(int* c, const int* a, const int* b, int matWidth) {
    __shared__ int tileA[TILE_SIZE][TILE_SIZE];
    __shared__ int tileB[TILE_SIZE][TILE_SIZE];

    int baseRow = blockIdx.x * TILE_SIZE;
    int baseCol = blockIdx.y * TILE_SIZE;
    int loop = matWidth / TILE_SIZE;
    int val = 0;
    for (int startMulIndex = 0; startMulIndex < 1024; startMulIndex += TILE_SIZE) {
        tileA[threadIdx.x][threadIdx.y] = a[(baseRow + threadIdx.x)       * matWidth + startMulIndex + threadIdx.y];
        tileB[threadIdx.x][threadIdx.y] = b[(startMulIndex + threadIdx.x) * matWidth + baseCol       + threadIdx.y];

        __syncthreads();
        for (int i = 0; i < TILE_SIZE; ++i) {
            val += tileA[threadIdx.x][i] * tileB[i][threadIdx.y];
        }
        __syncthreads();
    }
    c[(baseRow + threadIdx.x) * matWidth + baseCol + threadIdx.y] = val;
}

int main()
{
    int* a = new int[1024 * 1024];
    int* b = new int [1024 * 1024];
    int* c = new int [1024 * 1024];
    printf("Creating data\n");
    for (int row = 0; row < 1024; ++row) {
        for (int col = 0; col < 1024; ++col) {
            a[row * 1024 + col] = row > col ? row : col;
            b[row * 1024 + col] = row > col ? col : row;
            c[row * 1024 + col] = 0;
        }
    }
    matMulWithCuda(c, a, b);
    printf("Checking calculate result...\n");
    for (int row = 0; row < 1024; ++row) {
        for (int col = 0; col < 1024; ++col) {
            int val = 0;
            for (int i = 0; i < 1024; ++i) {
                val += a[row*1024+i] * b[i * 1024 + col];
            }
            if (val != c[row * 1024 + col]) {
                printf("Calculation wrong at %d %d %d != %d\n", row, col, c[row * 1024 + col], val);
                return -1;
            }
        }
    }
    printf("Correct calculation...\n");

    return 0;
}

hipError_t matMulWithCuda(int* c, const int* a, const int* b) {
    //Malloc
    hipError_t result = hipErrorUnknown;

    result = hipSetDevice(0);
    if (result != hipSuccess) {
        printf("Set device failed");
        return result;
    }

    int* devA, * devB, * devC;
    result = hipMalloc((void**)&devA, 1024 * 1024 * sizeof(int));
    if (result != hipSuccess) {
        printf("Malloc Mat A failed");
        return result;
    }
    result = hipMalloc((void**)&devB, 1024 * 1024 * sizeof(int));
    if (result != hipSuccess) {
        printf("Malloc Mat B failed");
        hipFree(devA);
        return result;
    }
    result = hipMalloc((void**)&devC, 1024 * 1024 * sizeof(int));
    if (result != hipSuccess) {
        printf("Malloc Mat C failed");
        hipFree(devA);
        hipFree(devB);
        return result;
    }
    if(result == hipSuccess) result = hipMemcpy(devA, a, 1024 * 1024 * sizeof(int), hipMemcpyHostToDevice);

    if(result != hipSuccess) printf("Input data failed for A %d\n", result);

    if(result == hipSuccess) result = hipMemcpy(devB, b, 1024 * 1024 * sizeof(int), hipMemcpyHostToDevice);

    if (result != hipSuccess) printf("Input data failed for B %d\n", result);

    if (result == hipSuccess) {
        dim3 gridDim(64, 64, 1);
        dim3 blockDim(16, 16, 1);
        printf("Start calculate \n");
        MatMul <<< gridDim, blockDim >>> (devC, devA, devB, 1024);
        printf("End calculate \n");
        result = hipMemcpy(c, devC, 1024 * 1024 * sizeof(int), hipMemcpyDeviceToHost);
        if (result != hipSuccess) {
            printf("Get calculation result failed\n");
        }
    }

    result = hipFree(devA);
    result = hipFree(devB);
    result = hipFree(devC);
    return result;
}